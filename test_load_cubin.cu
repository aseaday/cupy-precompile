
#include <hip/hip_runtime.h>

extern "C" __global__
void test_div(const float* x1, const float* x2, float* y, unsigned int N) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < N)
        y[tid] = x1[tid] / (x2[tid] + 1.0);
}
